#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <cstdlib>
#include <ctime>
#include <cstdint>


__global__ void reduceNoBankConflict(int* idata, int* odata) {
    __shared__ int data[32][32];

    uint tid = blockIdx.x * blockDim.x + threadIdx.x;
    uint row = threadIdx.x / 32;
    uint col = threadIdx.x % 32;
    data[row][col] = idata[tid];
    __syncthreads();

    for (int stride = 16; stride > 0; stride = stride >> 1) {
        if (row < stride)
            data[row][col] += data[row + stride][col];
        __syncthreads();
    }

    for (int stride = 16; stride > 0; stride = stride >> 1) {
        if (threadIdx.x < stride)
            data[0][col] += data[0][col + stride];
        __syncthreads();
    }

    if (threadIdx.x == 0) odata[blockIdx.x] = data[0][0];
}


__global__ void reduceUnroll(int* idata, int* odata) {
    __shared__ int data[32][32];

    uint tid = blockIdx.x * blockDim.x + threadIdx.x;
    uint row = threadIdx.x / 32;
    uint col = threadIdx.x % 32;
    data[row][col] = idata[tid];
    __syncthreads();

    for (int stride = 16; stride > 0; stride = stride >> 1) {
        if (row < stride)
            data[row][col] += data[row + stride][col];
        __syncthreads();
    }

    if (threadIdx.x < 32) {
        data[0][col] += data[0][col + 16];
        __syncthreads();
        data[0][col] += data[0][col + 8];
        __syncthreads();
        data[0][col] += data[0][col + 4];
        __syncthreads();
        data[0][col] += data[0][col + 2];
        __syncthreads();
        data[0][col] += data[0][col + 1];
        __syncthreads();
    }

    if (threadIdx.x == 0) odata[blockIdx.x] = data[0][0];
}

int reduceCPU(int* data, int size) {
    int sum = 0;
    for (int i{ 0 }; i < size; ++i) {
        sum += data[i];
    }
    return sum;
}

int main() {
    srand((unsigned)time(nullptr));

    const uint32_t size = 1 << 23;
    const uint32_t bytes = size * sizeof(int);
    int* Ic = (int*)malloc(bytes);
    for (int i{ 0 }; i < size; ++i) {
        Ic[i] = static_cast<int>(rand() % 10 + 1);
    }

    double start = seconds();
    int cpuRes = reduceCPU(Ic, size);
    double elaps = seconds() - start;
    printf("reduceCPU elaps %f sec\n", elaps);


    SelectGPUDevice(0, false);
    int* Id = nullptr;
    hipMalloc(&Id, bytes);
    hipMemcpy(Id, Ic, bytes, hipMemcpyHostToDevice);

    dim3 BD{1024};
    dim3 GD{size / BD.x};
    const int oBytes = GD.x * sizeof(int);
    int* Od = nullptr;
    hipMalloc(&Od, oBytes);
    hipMemset(Od, 0, oBytes);
    start = seconds();
    reduceNoBankConflict <<< GD, BD >>> (Id, Od);
    CHECK(hipDeviceSynchronize());
    elaps = seconds() - start;
    printf("reduceNoBankConflict <<< %d, %d >>> elaps %f sec\n", GD.x, BD.x, elaps);

    int* Oc = (int*)malloc(oBytes);
    memset(Oc, 0, oBytes);
    hipMemcpy(Oc, Od, oBytes, hipMemcpyDeviceToHost);
    int gpuRes = 0;
    for (int i{ 0 }; i < GD.x; ++i) {
        gpuRes += Oc[i];
    }
    if (cpuRes == gpuRes)
        printf("result match: %d\n", cpuRes);


    hipMemcpy(Id, Ic, bytes, hipMemcpyHostToDevice);
    hipMemset(Od, 0, oBytes);
    start = seconds();
    reduceUnroll <<< GD, BD >>> (Id, Od);
    CHECK(hipDeviceSynchronize());
    elaps = seconds() - start;
    printf("reduceUnroll <<< %d, %d >>> elaps %f sec\n", GD.x, BD.x, elaps);

    memset(Oc, 0, oBytes);
    hipMemcpy(Oc, Od, oBytes, hipMemcpyDeviceToHost);
    gpuRes = 0;
    for (int i{ 0 }; i < GD.x; ++i) {
        gpuRes += Oc[i];
    }
    if (cpuRes == gpuRes)
        printf("result match: %d\n", cpuRes);


    hipFree(&Id);
    hipFree(&Od);
    free(Ic);
    free(Oc);

    hipDeviceReset();
    return 0;
}
