#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ float devData;

__global__ void checkGlobalVariable() {
    printf("Device: the value of the global variable is %f\n", devData);
    devData += 2.0f;
}

int main() {
    float value = 3.14f;
    hipMemcpyToSymbol(HIP_SYMBOL(devData), &value, sizeof(float));
    printf("Host: copied %f to the global variable\n", value);

    checkGlobalVariable<<<1, 1>>>();

    hipMemcpyFromSymbol(&value, HIP_SYMBOL(devData), sizeof(float));
    printf("Host: the value changed by the kernel to %f\n", value);

    hipDeviceReset();
    return 0;
}
