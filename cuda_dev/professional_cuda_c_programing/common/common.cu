#include "common.h"

void SelectGPUDevice(int dev, bool promptName/* = true */) {
    hipDeviceProp_t prop;
    CHECK(hipGetDeviceProperties(&prop, dev));
    CHECK(hipSetDevice(dev));

    if (promptName)
        printf("GPU name: %s\n", prop.name);
}

