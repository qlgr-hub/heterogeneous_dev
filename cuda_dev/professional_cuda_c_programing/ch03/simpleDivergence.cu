#include <cstddef>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

#define CHECK(call) \
{\
    const hipError_t error = call; \
    if (error != hipSuccess) \
    {\
        printf("Error: %s:%d, ", __FILE__, __LINE__); \
        printf("code:%d, reason: %s\n", error, hipGetErrorString(error)); \
        exit(EXIT_FAILURE); \
    }\
}

__global__ void mathKernel1(float *c) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float a, b;
    a = b = 0.0f;

    if (tid % 2 == 0) {
        a = 100.0f;
    }
    else {
        b = 200.0f;
    }

    c [tid] = a + b;
}

__global__ void mathKernel2(float *c) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float a, b;
    a = b = 0.0f;

    if ( (tid / warpSize) % 2 == 0 ) {
        a = 100.0f;
    }
    else {
        b = 200.0f;
    }
    c[tid] = a + b;
}

__global__ void mathKernel3(float *c) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float a, b;
    a = b = 0.0f;

    bool ipred = (tid % 2 == 0);
    if ( ipred ) {
        a = 100.0f;
    }
    if (!ipred) {
        b = 200.0f;
    }
    c[tid] = a + b;
}

__global__ void mathKernel4(float *c)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;

    int itid = tid >> 5;

    if (itid & 0x01 == 0)
    {
        ia = 100.0f;
    }
    else
    {
        ib = 200.0f;
    }

    c[tid] = ia + ib;
}

__global__ void warmingup(float *c)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;

    if ((tid / warpSize) % 2 == 0)
    {
        ia = 100.0f;
    }
    else
    {
        ib = 200.0f;
    }

    c[tid] = ia + ib;
}

inline double seconds()
{
    struct timeval tp;
    struct timezone tzp;
    int i = gettimeofday(&tp, &tzp);
    return ((double)tp.tv_sec + (double)tp.tv_usec * 1.e-6);
}

int main(int argc, char **argv) {

    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // set up data size
    int size = 64;
    int blocksize = 64;
    if (argc > 1) blocksize = atoi(argv[1]);
    if (argc > 2) size      = atoi(argv[2]);
    printf("Data size %d \n", size);

    // set up execution configuration
    dim3 block{(uint)blocksize, 1};
    dim3 grid{(size + block.x - 1) / block.x, 1};
    printf("Execution Configure (block %d grid %d)\n", block.x, grid.x);

    // allocate gpu memory
    float *d_C;
    size_t nBytes = size * sizeof(float);
    CHECK(hipMalloc((float **)&d_C, nBytes));

    // run a warmup kernel to remove overhead
    double iStart, iElaps;
    CHECK(hipDeviceSynchronize());
    iStart = seconds();
    warmingup<<<grid, block>>>(d_C);
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;
    printf("warmup      <<< %4d %4d >>> elapsed %f sec \n", grid.x, block.x,
           iElaps );
    CHECK(hipGetLastError());

    // run kernel 1
    iStart = seconds();
    mathKernel1<<<grid, block>>>(d_C);
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;
    printf("mathKernel1 <<< %4d %4d >>> elapsed %f sec \n", grid.x, block.x,
           iElaps );
    CHECK(hipGetLastError());

    // run kernel 3
    iStart = seconds();
    mathKernel2<<<grid, block>>>(d_C);
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;
    printf("mathKernel2 <<< %4d %4d >>> elapsed %f sec \n", grid.x, block.x,
           iElaps );
    CHECK(hipGetLastError());

    // run kernel 3
    iStart = seconds();
    mathKernel3<<<grid, block>>>(d_C);
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;
    printf("mathKernel3 <<< %4d %4d >>> elapsed %f sec \n", grid.x, block.x,
           iElaps);
    CHECK(hipGetLastError());

    // run kernel 4
    iStart = seconds();
    mathKernel4<<<grid, block>>>(d_C);
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;
    printf("mathKernel4 <<< %4d %4d >>> elapsed %f sec \n", grid.x, block.x,
           iElaps);
    CHECK(hipGetLastError());

    // free gpu memory and reset divece
    CHECK(hipFree(d_C));
    CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}